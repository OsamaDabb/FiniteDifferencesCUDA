
#include <hip/hip_runtime.h>
#include<stdint.h>
#include<iostream>

int main(){
    int64_t nvtx_scale = ((int64_t)1)<<14;

    uint64_t* cost = (uint64_t*)malloc(sizeof(uint64_t)*nvtx_scale);

    for(int64_t i=0; i < nvtx_scale; i++)
        cost[i] = uint64_t(123456789);

    uint64_t* dcost;
    hipMalloc(&dcost, nvtx_scale*sizeof(uint64_t));
    hipMemcpy(dcost, cost, sizeof(uint64_t)*nvtx_scale, hipMemcpyHostToDevice);

    memset(cost, 0, sizeof(uint64_t)*nvtx_scale);
    hipMemcpy(cost, dcost, sizeof(uint64_t)*nvtx_scale, hipMemcpyDeviceToHost);

    for(int i=0; i<10; i++) {
        std::cout << i << " " << cost[i] << std::endl;
    }

    return 0;
}